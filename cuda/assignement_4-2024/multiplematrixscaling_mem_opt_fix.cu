#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h>  // Include OpenMP header

// Macro to check CUDA errors
#define CHECK_CUDA_ERROR(call)                                                   \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA Error: %s (error code: %d)\n",                 \
                    hipGetErrorString(err), err);                               \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

/*
TODO: topic to explore: 
    cudaMallocManaged
    cudaMemPrefetchAsync
    
To compile, profiling and running:
    compute --gpu
    module load cuda/11.0
    
    export OMP_NUM_THREADS=8 

    nvcc -Xptxas -lineinfo -v multiplematrixscaling_mem_opt_fix.cu -o multiplematrixscaling_mem_opt_fix -Xcompiler -fopenmp
    nvprof --export-profile multiplematrixscaling_mem_opt_fix.prof -f --analysis-metrics ./multiplematrixscaling_mem_opt_fix

    nvprof --output-profile multiplematrixscaling_mem_opt_fix.prof --metrics gld_efficiency,gst_efficiency,achieved_occupancy,warp_execution_efficiency -f --analysis-metrics ./multiplematrixscaling_mem_opt_fix

    nvprof --print-gpu-trace --csv ./multiplematrixscaling_mem_opt_fix > profile.csv


    ./multiplematrixscaling_mem_opt_fix
    
    sbatch multiplematrixscaling_mem_opt_fix.sh
    watch -n 1 squeue -u curso370
*/


// Optimized kernel
__global__ void matrix_scaling_factor_kernel_cuda(
    float * __restrict__ data,  // Input and output array; restrict keyword: avoid aliasing with any other pointer
    unsigned int N,             
    const float factor,         // leveraging const for optimization
    unsigned int repeat
) {
    __shared__ float tile[256];

    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int localIdx = threadIdx.x;

    if (globalIdx < N) {
        tile[localIdx] = data[globalIdx];
    }
    __syncthreads();

    if (globalIdx < N) {
        float val = tile[localIdx];
        for (unsigned int i = 0; i < repeat; i++) {
            val *= factor;
        }
        tile[localIdx] = val;
    }
    __syncthreads();


    if (globalIdx < N) {
        data[globalIdx] = tile[localIdx];
    }
}

int main(int argc, char *argv[]) {
    unsigned int N = 1 << 3; 
    unsigned int nn = N * N;
    unsigned int repeat = 2;
    float factors[] = {0.1, 0.2};
    unsigned int factorsLength = sizeof(factors) / sizeof(factors[0]);
    unsigned int sizeBytes = nn * sizeof(float);

    float *h_matrix, *h_results[factorsLength], *d_matrix[factorsLength];

    // Query device properties
    int device;
    CHECK_CUDA_ERROR(hipGetDevice(&device));

    int supportsUVA = 0;
    int supportsOverlap = 0;

    // Check for Unified Virtual Addressing (UVA) support
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&supportsUVA, hipDeviceAttributeUnifiedAddressing, device));
    printf("Device %d supports Unified Virtual Addressing (UVA): %s\n", device, supportsUVA ? "Yes" : "No");

    // Check for GPU Overlap support
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&supportsOverlap, hipDeviceAttributeConcurrentKernels, device));
    printf("Device %d supports GPU Overlap: %s\n", device, supportsOverlap ? "Yes" : "No");


    CHECK_CUDA_ERROR(hipHostMalloc(&h_matrix, sizeBytes, hipHostMallocDefault));

    #pragma omp parallel for
    for(unsigned int i = 0; i < N; i++){
        for(unsigned int j = 0; j < N; j++){
            unsigned int idx2d = i * N + j;
            h_matrix[idx2d] = i + j + 1;
        }
    } 

    // Allocate separate host buffers for results
    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipHostMalloc(&h_results[i], sizeBytes, hipHostMallocDefault));
    }

    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipMalloc(&d_matrix[i], sizeBytes));
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (nn + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t streams[factorsLength];
    hipEvent_t startk[factorsLength], endk[factorsLength];

    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
        CHECK_CUDA_ERROR(hipEventCreate(&startk[i]));
        CHECK_CUDA_ERROR(hipEventCreate(&endk[i]));
    }

    // Launch kernels and memory transfers concurrently
    for (unsigned int i = 0; i < factorsLength; i++) {
        printf("\nLaunching kernel for factor: %f, stream ID: %u", factors[i], i);

        // Asynchronous memory copy from host to device
        CHECK_CUDA_ERROR(hipMemcpyAsync(
            d_matrix[i], h_matrix, sizeBytes, hipMemcpyHostToDevice, streams[i]));

        // Record event before kernel launch
        CHECK_CUDA_ERROR(hipEventRecord(startk[i], streams[i]));

        // Launch kernel
        matrix_scaling_factor_kernel_cuda<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(
            d_matrix[i], nn, factors[i], repeat
        );

        // Record event after kernel execution
        CHECK_CUDA_ERROR(hipEventRecord(endk[i], streams[i]));

        // Asynchronous memory copy from device to separate host result buffer
        CHECK_CUDA_ERROR(hipMemcpyAsync(
            h_results[i], d_matrix[i], sizeBytes, hipMemcpyDeviceToHost, streams[i]));
    }

    // Wait for all streams to finish and collect statistics
    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipStreamSynchronize(streams[i]));
        float elapsedTime = 0.0f;
        CHECK_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, startk[i], endk[i]));
        printf("\nKernel for factor: %f; Elapsed time: %f ms", factors[i], elapsedTime);
    }

    #pragma omp parallel for
    for (unsigned int i = 0; i < nn; i++) {
        h_matrix[i] = 0.0f;
        for (unsigned int j = 0; j < factorsLength; j++) {
            h_matrix[i] += h_results[j][i];
        }
    }

    printf("\n\nUpdated Matrix:\n");
    #pragma omp parallel for
    for (unsigned int i = 0; i < N; i++) {
        for (unsigned int j = 0; j < N; j++) {
            unsigned int idx2d = i * N + j;
            #pragma omp critical
            printf("%f, ", h_matrix[idx2d]);
        }
        #pragma omp critical
        printf("\n");
    }

    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
        CHECK_CUDA_ERROR(hipEventDestroy(startk[i]));
        CHECK_CUDA_ERROR(hipEventDestroy(endk[i]));
        CHECK_CUDA_ERROR(hipFree(d_matrix[i]));
        CHECK_CUDA_ERROR(hipHostFree(h_results[i]));
    }
    CHECK_CUDA_ERROR(hipHostFree(h_matrix));

    CHECK_CUDA_ERROR(hipDeviceReset());

    return 0;
}
