#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h>  // Include OpenMP header

#define CHECK_CUDA_ERROR(call)                                                   \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            fprintf(stderr, "CUDA Error: %s (error code: %d)\n",                 \
                    hipGetErrorString(err), err);                               \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

// Optimized kernel
__global__ void matrix_scaling_factor_kernel_cuda(
    float * __restrict__ data,
    unsigned int N,
    const float factor,
    unsigned int repeat
) {
    __shared__ float tile[256];

    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int localIdx = threadIdx.x;

    if (globalIdx < N) {
        tile[localIdx] = data[globalIdx];
    }
    __syncthreads();

    if (globalIdx < N) {
        float val = tile[localIdx];
        for (unsigned int i = 0; i < repeat; i++) {
            val *= factor;
        }
        tile[localIdx] = val;
    }
    __syncthreads();

    if (globalIdx < N) {
        data[globalIdx] = tile[localIdx];
    }
}

int main(int argc, char *argv[]) {
    unsigned int N = 1 << 3;
    unsigned int nn = N * N;
    unsigned int repeat = 2;
    float factors[] = {0.1, 0.2};
    unsigned int factorsLength = sizeof(factors) / sizeof(factors[0]);
    unsigned int sizeBytes = nn * sizeof(float);

    float *matrix, *results[factorsLength];

    // Query device properties
    int device;
    CHECK_CUDA_ERROR(hipGetDevice(&device));

    int supportsUVA = 0;
    int supportsOverlap = 0;

    // Check for Unified Virtual Addressing (UVA) support
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&supportsUVA, hipDeviceAttributeUnifiedAddressing, device));
    printf("Device %d supports Unified Virtual Addressing (UVA): %s\n", device, supportsUVA ? "Yes" : "No");

    // Check for GPU Overlap support
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&supportsOverlap, hipDeviceAttributeConcurrentKernels, device));
    printf("Device %d supports GPU Overlap: %s\n", device, supportsOverlap ? "Yes" : "No");

    CHECK_CUDA_ERROR(hipMallocManaged(&matrix, sizeBytes));
    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipMallocManaged(&results[i], sizeBytes));
    }

    // Initialize matrix with OpenMP
    #pragma omp parallel for
    for (unsigned int i = 0; i < nn; i++) {
        matrix[i] = (float)(i + 1);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (nn + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t streams[factorsLength];
    hipEvent_t startk[factorsLength], endk[factorsLength];

    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
        CHECK_CUDA_ERROR(hipEventCreate(&startk[i]));
        CHECK_CUDA_ERROR(hipEventCreate(&endk[i]));
    }

    // Launch kernels with unified memory prefetching
    for (unsigned int i = 0; i < factorsLength; i++) {
        printf("\nLaunching kernel for factor: %f, stream ID: %u", factors[i], i);

        // Prefetch memory to the GPU for current stream
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(matrix, sizeBytes, device, streams[i]));
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(results[i], sizeBytes, device, streams[i]));

        // Record event before kernel launch
        CHECK_CUDA_ERROR(hipEventRecord(startk[i], streams[i]));

        // Launch kernel
        matrix_scaling_factor_kernel_cuda<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(
            matrix, nn, factors[i], repeat
        );

        // Record event after kernel execution
        CHECK_CUDA_ERROR(hipEventRecord(endk[i], streams[i]));

        // Prefetch result back to the host
        CHECK_CUDA_ERROR(hipMemPrefetchAsync(results[i], sizeBytes, hipCpuDeviceId, streams[i]));
    }

    // Wait for all streams to finish and collect statistics
    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipStreamSynchronize(streams[i]));
        float elapsedTime = 0.0f;
        CHECK_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, startk[i], endk[i]));
        printf("\nKernel for factor: %f; Elapsed time: %f ms", factors[i], elapsedTime);
    }

    #pragma omp parallel for
    for (unsigned int i = 0; i < nn; i++) {
        matrix[i] = 0.0f;
        for (unsigned int j = 0; j < factorsLength; j++) {
            matrix[i] += results[j][i];
        }
    }

    printf("\n\nUpdated Matrix:");
    #pragma omp parallel for
    for (unsigned int i = 0; i < N; i++) {
        for (unsigned int j = 0; j < N; j++) {
            unsigned int idx2d = i * N + j;
            #pragma omp critical
            printf("%f, ", matrix[idx2d]);
        }
        #pragma omp critical
        printf("\n");
    }

    for (unsigned int i = 0; i < factorsLength; i++) {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
        CHECK_CUDA_ERROR(hipEventDestroy(startk[i]));
        CHECK_CUDA_ERROR(hipEventDestroy(endk[i]));
        CHECK_CUDA_ERROR(hipFree(results[i]));
    }
    CHECK_CUDA_ERROR(hipFree(matrix));

    CHECK_CUDA_ERROR(hipDeviceReset());
    
    return 0;
}
