#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define CHECK_CUDA_ERROR(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error: %s (code %d)\n", hipGetErrorString(err), err); \
            exit(EXIT_FAILURE); \
        } \
    }

__device__ void merge(int* data, int* temp, int left, int mid, int right) {
    int i = left, j = mid + 1, k = left;
    while (i <= mid && j <= right) {
        if (data[i] <= data[j]) temp[k++] = data[i++];
        else temp[k++] = data[j++];
    }
    while (i <= mid) temp[k++] = data[i++];
    while (j <= right) temp[k++] = data[j++];
    for (i = left; i <= right; i++) data[i] = temp[i];
}

__global__ void sort_kernel(int* data, int size) {
    extern __shared__ int temp[];
    for (int width = 1; width < size; width *= 2) {
        for (int i = 0; i < size; i += 2 * width) {
            int left = i;
            int mid = min(i + width - 1, size - 1);
            int right = min(i + 2 * width - 1, size - 1);
            if (mid < right)
                merge(data, temp, left, mid, right);
        }
    }
}

__global__ void merge_kernel(const int* left, int sizeL, const int* right, int sizeR, int* output) {
    int i = 0, j = 0, k = 0;
    while (i < sizeL && j < sizeR) {
        output[k++] = (left[i] <= right[j]) ? left[i++] : right[j++];
    }
    while (i < sizeL) output[k++] = left[i++];
    while (j < sizeR) output[k++] = right[j++];
}

void merge_cpu(int* arr, int l, int m, int r, int* temp) {
    int i = l, j = m + 1, k = l;
    while (i <= m && j <= r) {
        if (arr[i] <= arr[j]) temp[k++] = arr[i++];
        else temp[k++] = arr[j++];
    }
    while (i <= m) temp[k++] = arr[i++];
    while (j <= r) temp[k++] = arr[j++];
    for (i = l; i <= r; ++i) arr[i] = temp[i];
}

void merge_sort_cpu_parallel(int* arr, int l, int r, int* temp, int depth) {
    if (l < r) {
        int m = (l + r) / 2;
        if (depth <= 0) {
            merge_sort_cpu_parallel(arr, l, m, temp, depth - 1);
            merge_sort_cpu_parallel(arr, m + 1, r, temp, depth - 1);
        } else {
            #pragma omp parallel sections
            {
                #pragma omp section
                merge_sort_cpu_parallel(arr, l, m, temp, depth - 1);
                #pragma omp section
                merge_sort_cpu_parallel(arr, m + 1, r, temp, depth - 1);
            }
        }
        merge_cpu(arr, l, m, r, temp);
    }
}

void check_result_cpu(const int* original, const int* sorted, int size) {
    int* reference = (int*)malloc(size * sizeof(int));
    int* temp = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) reference[i] = original[i];

    merge_sort_cpu_parallel(reference, 0, size - 1, temp, 3);

    int errors = 0;
    for (int i = 0; i < size; i++) {
        if (reference[i] != sorted[i]) {
            printf("Mismatch at idx %d: CPU = %d, GPU = %d\n", i, reference[i], sorted[i]);
            errors++;
        }
    }

    if (errors == 0)
        printf("CPU-GPU checked OK\n");
    else
        printf("CPU-GPU N-OK, with %d mismatches.\n", errors);

    free(reference);
    free(temp);
}

int main() {
    const int N = 16;
    int h_input[N] = {20, 5, 3, 9, 1, 4, 7, 6, 14, 13, 11, 10, 2, 8, 12, 15};
    int *h_output;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_output, N * sizeof(int), hipHostMallocDefault));

    int mid = N / 2;
    int *d_left, *d_right, *d_merged;
    CHECK_CUDA_ERROR(hipMalloc(&d_left, mid * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_right, (N - mid) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_merged, N * sizeof(int)));

    hipStream_t s_left, s_right, s_merge;
    CHECK_CUDA_ERROR(hipStreamCreate(&s_left));
    CHECK_CUDA_ERROR(hipStreamCreate(&s_right));
    CHECK_CUDA_ERROR(hipStreamCreate(&s_merge));

    CHECK_CUDA_ERROR(hipMemcpyAsync(d_left, h_input, mid * sizeof(int), hipMemcpyHostToDevice, s_left));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_right, h_input + mid, (N - mid) * sizeof(int), hipMemcpyHostToDevice, s_right));

    //to sort each half
    sort_kernel<<<1, 1, mid * sizeof(int), s_left>>>(d_left, mid);
    sort_kernel<<<1, 1, (N - mid) * sizeof(int), s_right>>>(d_right, N - mid);

    hipEvent_t eL, eR;
    CHECK_CUDA_ERROR(hipEventCreate(&eL));
    CHECK_CUDA_ERROR(hipEventCreate(&eR));
    CHECK_CUDA_ERROR(hipEventRecord(eL, s_left));
    CHECK_CUDA_ERROR(hipEventRecord(eR, s_right));
    CHECK_CUDA_ERROR(hipStreamWaitEvent(s_merge, eL, 0));
    CHECK_CUDA_ERROR(hipStreamWaitEvent(s_merge, eR, 0));

    // Merge result
    merge_kernel<<<1, 1, 0, s_merge>>>(d_left, mid, d_right, N - mid, d_merged);

    // Copy back
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_merged, N * sizeof(int), hipMemcpyDeviceToHost, s_merge));
    CHECK_CUDA_ERROR(hipStreamSynchronize(s_merge));

    printf("GPU Sorted: ");
    for (int i = 0; i < N; ++i) printf("%d ", h_output[i]);
    printf("\n");

    check_result_cpu(h_input, h_output, N);

    // Cleanup
    hipHostFree(h_output);
    hipFree(d_left);
    hipFree(d_right);
    hipFree(d_merged);
    hipStreamDestroy(s_left);
    hipStreamDestroy(s_right);
    hipStreamDestroy(s_merge);
    hipEventDestroy(eL);
    hipEventDestroy(eR);
 
    CHECK_CUDA_ERROR(hipDeviceReset());
    printf("CUDA device reset.\n");

    return 0;
}